
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>


__global__ void polynomial_expansion (float* poly, int degree, int n, float* array) {
  float out = 0.;
  float xdegree = 1.;
  
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n){
    float x = array[i];
    for (int k=0; k<=degree; ++k) {
      out += xdegree*poly[k];
      xdegree *= x;
    }
    array[i] = out;
  }


}

int main (int argc, char* argv[]) {
  //TODO: add usage
  
  if (argc < 3) {
     std::cerr<<"usage: "<<argv[0]<<" n degree"<<std::endl;
     return -1;
  }

  int n = atoi(argv[1]); 
  int degree = atoi(argv[2]);
  int nbiter = 1;

  float* array = new float[n];
  float* poly = new float[degree+1];
  float *xPointer, *coeffPointer; 
  
  for (int i=0; i<n; ++i)
    array[i] = 1.0;

  for (int i=0; i<degree+1; ++i)
    poly[i] = 1.0;

  
  hipMalloc(&xPointer, n*sizeof(float)); 
  hipMalloc(&coeffPointer, (degree+1)*sizeof(float)); 
 
  hipMemcpy(xPointer, array, n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(coeffPointer, poly, (degree+1)*sizeof(float), hipMemcpyHostToDevice); 
  std::chrono::time_point<std::chrono::system_clock> begin, end;
  begin = std::chrono::system_clock::now();
  
  
  for (int iter = 0; iter<nbiter; ++iter){
     polynomial_expansion<<<(n+255)/256, 256>>>(coeffPointer, degree, n, xPointer);
  }
  hipDeviceSynchronize();
  
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) 
    printf("Error: %s\n", hipGetErrorString(err)); 

  hipMemcpy(array, xPointer, n*sizeof(float), hipMemcpyDeviceToHost);
  
  end = std::chrono::system_clock::now();
  std::chrono::duration<double> totaltime = (end-begin)/nbiter;

  std::cerr<<array[0]<<std::endl;
  std::cout<<n<<" "<<degree<<" "<<totaltime.count()<<std::endl;

  delete[] array;
  delete[] poly;
  hipFree(xPointer); 
  hipFree(coeffPointer);
  return 0;
}